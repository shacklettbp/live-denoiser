#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "bind_buffers.h"

#define CU_CHECK_THROW(x)                                                                    \
  do {                                                                                       \
    hipError_t result = x;                                                                     \
    if (result != hipSuccess) {                                                            \
      const char *msg;                                                                       \
      hipDrvGetErrorName(result, &msg);                                                          \
      throw std::runtime_error(std::string("CUDA Error: " #x " failed with error ") + msg);  \
    }                                                                                        \
  } while(0)

#define CUDA_CHECK_THROW(x)                                                                                          \
  do {                                                                                                               \
    hipError_t result = x;                                                                                          \
    if (result != hipSuccess)                                                                                       \
      throw std::runtime_error(std::string("CUDA Error: " #x " failed with error ") + hipGetErrorString(result));   \
  } while(0)

std::pair<void *, void *> getDevicePtr(void *handle, uint64_t bytes)
{
    hipExternalMemoryHandleDesc externalMemoryHandleDesc;
    memset(&externalMemoryHandleDesc, 0, sizeof(externalMemoryHandleDesc));

    externalMemoryHandleDesc.type = hipExternalMemoryHandleTypeD3D12Resource;
    externalMemoryHandleDesc.handle.win32.handle = handle;
    externalMemoryHandleDesc.size = bytes;
    externalMemoryHandleDesc.flags = hipExternalMemoryDedicated;

    hipExternalMemory_t externalMemory;
    CUDA_CHECK_THROW(hipImportExternalMemory(&externalMemory, &externalMemoryHandleDesc));

    hipExternalMemoryBufferDesc bufferDesc;
    bufferDesc.offset = 0;
    bufferDesc.size = externalMemoryHandleDesc.size;
    bufferDesc.flags = 0;

    void *dev_ptr;
    CUDA_CHECK_THROW(hipExternalMemoryGetMappedBuffer(&dev_ptr, externalMemory, &bufferDesc));

    return std::make_pair(dev_ptr, externalMemory);
}
